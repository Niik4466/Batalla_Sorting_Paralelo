#include <hip/hip_runtime.h>
#include <iostream>

int main(){
  int device, sm_count;
  hipGetDevice(&device);
  hipDeviceGetAttribute(&sm_count, hipDeviceAttributeMultiprocessorCount, device);
  printf("%d", sm_count);
  return 0;
}
